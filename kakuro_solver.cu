
#include <hip/hip_runtime.h>
#include <iostream>
#include <string>

#include <fstream>
#include <sstream>
#include <vector>

#include <bits/stdc++.h>
#include <array>

using namespace std;

enum direction
{
  d_down,
  d_right,
  none
};

#define COORD std::pair<int, int>

// #define DEBUG

int iter = 0;

//////////////////////////////////////////////
// Auxiliary functions for preparing problem //
//////////////////////////////////////////////

void display_arr(int *arr, int n)
{

  cout << "arr: ";

  for (int i = 0; i < n; i++)
  {
    cout << arr[i] << " ";
  }

  cout << endl;
}

void print_coords(COORD start, COORD end)
{

  cout << "Start:" << start.first << "," << start.second << endl;
  cout << "End:" << end.first << "," << end.second << endl;
}

int find_length(COORD start, COORD end, direction dir)
{

  if (dir == d_down)
    return end.first - start.first;
  if (dir == d_right)
    return end.second - start.second;

  return -1;
}

void convert_sol(int **mat, int **&sol_mat, int m, int n)
{

  sol_mat = new int *[m]; // Rows
  for (int i = 0; i < m; i++)
  {
    sol_mat[i] = new int[n]; // Cols
  }

  for (int i = 0; i < m; i++)
  {
    for (int j = 0; j < m; j++)
    {
      if (mat[i][j] == -2)
        sol_mat[i][j] = -2; // Empty value cell
      else
        sol_mat[i][j] = -1; // Hint or empty cell
    }
  }
}

void print_one_matrix(int **matrix, int m, int n)
{
  std::cout << "Matrix: " << std::endl;
  for (int i = 0; i < m; i++)
  { // rows
    for (int j = 0; j < n; j++)
    { // cols
      std::cout << matrix[i][j] << "\t";
    }
    std::cout << "\n";
  }
}

/// Auxiliary functions

struct sum
{
  COORD start;
  COORD end;

  int hint;
  int dir;
  int length;
  int *arr;

  void print_sum()
  {
    cout << "############################" << endl;
    cout << "Creating sum with: " << endl;
    print_coords(start, end);
    cout << "Hint: " << hint << endl;
    cout << "Direction: " << dir << endl;
    cout << "Length: " << length << endl;
    cout << "############################" << endl;
  }

  sum(COORD _start, COORD _end, int _hint, direction _dir) : start(_start), end(_end), hint(_hint), dir(_dir)
  {
    length = find_length(_start, _end, _dir);
    arr = new int[length];
#ifdef DEBUG
    cout << "############################" << endl;
    cout << "Creating sum with: " << endl;
    print_coords(start, end);
    cout << "Hint: " << hint << endl;
    cout << "Direction: " << dir << endl;
    cout << "Length: " << length << endl;
    cout << "############################" << endl;
#endif
  }

  //~sum(){
  // delete arr;
  //}
};

COORD find_end(int **matrix, int m, int n, int i, int j, direction dir)
{ // 0 down 1 right

  if (dir == d_right)
  {
    for (int jj = j + 1; jj < n; jj++)
    {
      if (matrix[i][jj] != -2 || jj == n - 1)
      {
        if (matrix[i][jj] == -2 && jj == n - 1)
          jj++;
        COORD END = COORD(i, jj);
        return END;
      }
    }
  }

  if (dir == d_down)
  {
    for (int ii = i + 1; ii < m; ii++)
    {
      if (matrix[ii][j] != -2 || ii == m - 1)
      {
        if (matrix[ii][j] == -2 && ii == m - 1)
          ii++;
        COORD END = COORD(ii, j);
        return END;
      }
    }
  }
  return;
}

vector<sum> get_sums(int **matrix, int m, int n)
{

  vector<sum> sums;

  for (int i = 0; i < m; i++)
  {
    for (int j = 0; j < n; j++)
    {
      int val = matrix[i][j];
      if (val != -1 && val != -2)
      {
        int hint = val;
        hint = hint / 10;

        if ((hint % 100) == 0)
        {
          hint = (int)(hint / 100);
          COORD START = COORD(i, j + 1);
          COORD END = find_end(matrix, m, n, i, j, d_right);
          sum _sum = sum(START, END, hint, d_right);
          sums.push_back(_sum);
        }

        else
        {
          int div = (int)(hint / 100);
          int rem = (int)(hint % 100);

          if (div == 0 && rem != 0)
          {
            COORD START = COORD(i + 1, j);
            COORD END = find_end(matrix, m, n, i, j, d_down);
            sum _sum = sum(START, END, rem, d_down);
            sums.push_back(_sum);
          }

          if (div != 0 && rem != 0)
          {
            COORD START1 = COORD(i + 1, j);
            COORD START2 = COORD(i, j + 1);
            COORD END1 = find_end(matrix, m, n, i, j, d_down);
            COORD END2 = find_end(matrix, m, n, i, j, d_right);
            sum _sum1 = sum(START1, END1, rem, d_down);
            sum _sum2 = sum(START2, END2, div, d_right);
            sums.push_back(_sum1);
            sums.push_back(_sum2);
          }
        }
      }
    }
  }
  return sums;
}

void read_matrix(int **&matrix, std::ifstream &afile, int m, int n)
{

  matrix = new int *[m]; // rows

  for (int i = 0; i < m; i++)
  {
    matrix[i] = new int[n]; // cols
  }

  int val;
  for (int i = 0; i < m; i++)
  {
    for (int j = 0; j < n; j++)
    {
      afile >> val;
      matrix[i][j] = val;
    }
  }
}

void sol_to_file(int **mat, int **sol_mat, int m, int n)
{

  string fname = "visualize.kakuro";
  ofstream to_write(fname);

  to_write << m << " " << n << "\n";

  for (int i = 0; i < m; i++)
  {
    for (int j = 0; j < n; j++)
    {
      if (mat[i][j] != -2)
        to_write << mat[i][j] << " ";
      else
        to_write << sol_mat[i][j] << " ";
    }
    to_write << "\n";
  }

  to_write.close();
}

//////////////////////////////////////////////
// Auxiliary functions for preparing problem //
//////////////////////////////////////////////

///////////////////////////////////////////////////
// Auxiliary functions for preparing CUDA setting //
///////////////////////////////////////////////////

void flatten_sums(vector<sum> sums, int *h_sum_starts_x, int *h_sum_starts_y, int *h_sum_ends_x, int *h_sum_ends_y, int *h_sum_hints, int *h_sum_lengths, int *h_sum_dirs, int no_sums)
{

  for (int i = 0; i < no_sums; i++)
  {

    h_sum_starts_x[i] = sums[i].start.first;
    h_sum_starts_y[i] = sums[i].start.second;

    h_sum_ends_x[i] = sums[i].end.first;
    h_sum_ends_y[i] = sums[i].end.second;

    h_sum_hints[i] = sums[i].hint;
    h_sum_lengths[i] = sums[i].length;

    h_sum_dirs[i] = sums[i].dir;
  }
}

void print_flattened(int *h_sum_starts_x, int *h_sum_starts_y, int *h_sum_ends_x, int *h_sum_ends_y, int *h_sum_hints, int *h_sum_lengths, int *h_sum_dirs, int no_sums)
{

  cout << "###h_sum_starts_x: " << endl;
  for (int i = 0; i < no_sums; i++)
  {
    cout << h_sum_starts_x[i] << " ";
  }
  cout << endl;

  cout << "###h_sum_starts_y: " << endl;
  for (int i = 0; i < no_sums; i++)
  {
    cout << h_sum_starts_y[i] << " ";
  }
  cout << endl;

  cout << "###h_sum_ends_x: " << endl;
  for (int i = 0; i < no_sums; i++)
  {
    cout << h_sum_ends_x[i] << " ";
  }
  cout << endl;

  cout << "###h_sum_ends_y: " << endl;
  for (int i = 0; i < no_sums; i++)
  {
    cout << h_sum_ends_y[i] << " ";
  }
  cout << endl;

  cout << "###h_sum_hints: " << endl;
  for (int i = 0; i < no_sums; i++)
  {
    cout << h_sum_hints[i] << " ";
  }
  cout << endl;

  cout << "###h_sum_lengths: " << endl;
  for (int i = 0; i < no_sums; i++)
  {
    cout << h_sum_lengths[i] << " ";
  }
  cout << endl;

  cout << "###h_sum_dirs: " << endl;
  for (int i = 0; i < no_sums; i++)
  {
    cout << h_sum_dirs[i] << " ";
  }
  cout << endl;
}

void flatten_sol_mat(int **sol_mat, int *h_sol_mat, int m, int n)
{

  for (int i = 0; i < m; i++)
  {
    for (int j = 0; j < n; j++)
    {
      h_sol_mat[i * n + j] = sol_mat[i][j];
    }
  }
}

void print_flattened_matrix(int *h_sol_mat, int m, int n)
{

  cout << "###Flattened matrix: " << endl;
  for (int i = 0; i < m; i++)
  {
    for (int j = 0; j < n; j++)
    {
      cout << h_sol_mat[i * n + j] << " ";
    }
    cout << endl;
  }
  cout << endl;
}

///////////////////////////////////////////////////
// Auxiliary functions for preparing CUDA setting //
///////////////////////////////////////////////////

///////////////////
// CUDA FUNCTIONS //
///////////////////

__device__ bool check_solution(int *d_sol_mat, int m, int n, int *d_sum_starts_x, int *d_sum_starts_y, int *d_sum_ends_x, int *d_sum_ends_y, int *d_sum_hints, int *d_sum_lengths, int *d_sum_dirs, int no_sums)
{
  for (int i = 0; i < no_sums; ++i)
  {
    int sum = 0;
    //   printf("Checking sum: %d\n", i);
    // printf("Starts: x = %d, y = %d\n", d_sum_starts_x[i], d_sum_starts_y[i]);
    // printf("Ends: x = %d, y = %d\n", d_sum_ends_x[i], d_sum_ends_y[i]);
    if (d_sum_dirs[i] == d_down)
    {
      for (int row = d_sum_starts_x[i]; row < d_sum_ends_x[i]; ++row)
      {

        sum += d_sol_mat[row * n + d_sum_starts_y[i]];
        // printf("Down direction: row = %d, sum = %d\n", row, sum);
      }
    }
    else if (d_sum_dirs[i] == d_right)
    {
      for (int col = d_sum_starts_y[i]; col < d_sum_ends_y[i]; ++col)
      {

        sum += d_sol_mat[d_sum_starts_x[i] * n + col];
        //    printf("Right direction: col = %d, sum = %d\n", col, sum);
      }
    }
     printf("Sum hint: %d, calculated sum: %d\n", d_sum_hints[i], sum);
    if (sum != d_sum_hints[i])
    {

      return false;
    }
  }
  return true;
}

__global__ void kakuro_kernel(int *d_sum_starts_x, int *d_sum_starts_y, int *d_sum_ends_x, int *d_sum_ends_y,
                              int *d_sum_hints, int *d_sum_lengths, int *d_sum_dirs, int *d_sol_mat, int *d_perms, int *d_t_mats, int m, int n, int no_sums, volatile bool *solved)
{
  // Compute unique thread index
  int index = threadIdx.x + blockIdx.x * blockDim.x;

  // Each thread works with a unique permutation of the numbers 1 to 9
  int *perm = &d_perms[index * no_sums];
  //  printf("Thread index %d, permutation: ", index);
  for (int i = 0; i < no_sums; ++i)
  {
    //  printf("%d ", perm[i]);
  }
  int *local_sol_mat = &d_t_mats[index * m * n];

  // Copy the initial puzzle board into the local copy
  for (int i = 0; i < m * n; ++i)
  {

    local_sol_mat[i] = d_sol_mat[i];
  }

  // Insert the permutation into the empty cells in the puzzle
  for (int i = 0, row = 0, col = 0; i < no_sums && row < m; ++row)
  {
    for (col = 0; col < n && i < no_sums; ++col)
    {
      if (local_sol_mat[row * n + col] == -2)
      {                                           // If the cell is empty
        local_sol_mat[row * n + col] = perm[i++]; // Insert the i-th number of the permutation into the cell
      }
    }
  }
  // Print local_sol_mat

  // printf("here4");
  //  After inserting the permutation, check if it's a valid solution

  if (check_solution(local_sol_mat, m, n, d_sum_starts_x, d_sum_starts_y, d_sum_ends_x, d_sum_ends_y, d_sum_hints, d_sum_lengths, d_sum_dirs, no_sums))
  {
    *solved = true; // Update the solved flag
                    // This thread has finished
                    // If it's a valid solution, copy the local_sol_mat back to d_sol_mat
    printf("solution found");
    for (int i = 0; i < m * n; i++)
    {
      d_sol_mat[i] = local_sol_mat[i];
    }
    printf("Solution Matrix:\n");
    for (int row = 0; row < m; ++row)
    {
      for (int col = 0; col < n; ++col)
      {
        printf("%d ", d_sol_mat[row * n + col]);
      }
      printf("\n");
    }
    return; //
  }
  //  printf("no solution");
}

void generatePermutations(vector<int> &combination, vector<vector<int>> &perms)
{
  sort(combination.begin(), combination.end());
  do
  {
    perms.push_back(combination);
  } while (next_permutation(combination.begin(), combination.end()));
}

void permute(int a[], int n, int r, vector<vector<int>> &perms)
{
  vector<bool> v(n);
  fill(v.begin(), v.begin() + r, true);
  do
  {
    vector<int> combination;
    for (int i = 0; i < n; i++)
    {
      if (v[i])
      {
        combination.push_back(a[i]);
      }
    }
    generatePermutations(combination, perms);
  } while (prev_permutation(v.begin(), v.end()));
   
}

///////////////////
// CUDA FUNCTIONS //
///////////////////

int main(int argc, char **argv)
{

  std::string filename(argv[1]);
  std::ifstream file;
  file.open(filename.c_str());

  int m, n;
  file >> m;
  file >> n;

  int **mat;
  read_matrix(mat, file, m, n);
  print_one_matrix(mat, m, n);

  int **sol_mat;
  convert_sol(mat, sol_mat, m, n);
  // print_one_matrix(sol_mat, m, n);

  vector<sum> sums = get_sums(mat, m, n);

  // CUDA
  hipDeviceProp_t prop;
  hipGetDeviceProperties(&prop, 0);
  printf("==prop== Running on device: %d -- %s \n", 0, prop.name);
  printf("==prop== #of SM -- %d \n", prop.multiProcessorCount);
  printf("==prop== Max Threads Per Block: -- %d \n", prop.maxThreadsPerBlock);
  // Generate all permutations of size `no_sums`
int no_sums = sums.size();
int a[9] = {1, 2, 3, 4, 5, 6, 7, 8, 9};
  vector<vector<int>> perms;
  permute(a, 9, no_sums, perms);
int max_threads_per_block = 1024;
int total_permutations = perms.size();

int block_dim = min(total_permutations, max_threads_per_block);
int grid_dim = (total_permutations + block_dim - 1) / block_dim;
 // To D
 // To D
 cout<<"Block dimension: "<<block_dim<<endl;
 cout<<"total threads: "<< block_dim*grid_dim<<endl;

  

  // Flattening sums and matrix
  int *h_sum_starts_x = new int[no_sums];
  int *h_sum_starts_y = new int[no_sums];
  int *h_sum_ends_x = new int[no_sums];
  int *h_sum_ends_y = new int[no_sums];
  int *h_sum_hints = new int[no_sums];
  int *h_sum_lengths = new int[no_sums];
  int *h_sum_dirs = new int[no_sums];

  flatten_sums(sums, h_sum_starts_x, h_sum_starts_y, h_sum_ends_x, h_sum_ends_y, h_sum_hints, h_sum_lengths, h_sum_dirs, no_sums);

  print_flattened(h_sum_starts_x, h_sum_starts_y, h_sum_ends_x, h_sum_ends_y, h_sum_hints, h_sum_lengths, h_sum_dirs, no_sums);

  int *h_sol_mat;
  h_sol_mat = new int[m * n];

  flatten_sol_mat(sol_mat, h_sol_mat, m, n);

  print_flattened_matrix(h_sol_mat, m, n);

  // Declare device pointers and copy data into device
  int *d_sum_starts_x, *d_sum_starts_y, *d_sum_ends_x, *d_sum_ends_y, *d_sum_hints, *d_sum_lengths, *d_sum_dirs, *d_sol_mat, *d_t_mats;

  hipMalloc(&d_sum_starts_x, no_sums * sizeof(int));
  hipMalloc(&d_sum_starts_y, no_sums * sizeof(int));
  hipMalloc(&d_sum_ends_x, no_sums * sizeof(int));
  hipMalloc(&d_sum_ends_y, no_sums * sizeof(int));
  hipMalloc(&d_sum_hints, no_sums * sizeof(int));
  hipMalloc(&d_sum_lengths, no_sums * sizeof(int));
  hipMalloc(&d_sum_dirs, no_sums * sizeof(int));
  hipMalloc(&d_sol_mat, (m * n) * sizeof(int));
  hipMalloc(&d_t_mats, (m * n * grid_dim * block_dim) * sizeof(int)); // Allocating invidual matrix for each GPU thread
  // You may use this array if you will implement a thread-wise solution

  hipMemcpy(d_sum_starts_x, h_sum_starts_x, no_sums * sizeof(int), hipMemcpyHostToDevice);
  hipMemcpy(d_sum_starts_y, h_sum_starts_y, no_sums * sizeof(int), hipMemcpyHostToDevice);
  hipMemcpy(d_sum_ends_x, h_sum_ends_x, no_sums * sizeof(int), hipMemcpyHostToDevice);
  hipMemcpy(d_sum_ends_y, h_sum_ends_y, no_sums * sizeof(int), hipMemcpyHostToDevice);
  hipMemcpy(d_sum_hints, h_sum_hints, no_sums * sizeof(int), hipMemcpyHostToDevice);
  hipMemcpy(d_sum_lengths, h_sum_lengths, no_sums * sizeof(int), hipMemcpyHostToDevice);
  hipMemcpy(d_sum_dirs, h_sum_dirs, no_sums * sizeof(int), hipMemcpyHostToDevice);
  hipMemcpy(d_sol_mat, h_sol_mat, (m * n) * sizeof(int), hipMemcpyHostToDevice);
  // cout<<"matrix"<<endl;

  bool *solved = new bool; // Allocate memory for a bool variable
  *solved = false;
  bool *d_solved;

  hipMalloc(&d_solved, sizeof(bool));
  hipMemcpy(d_solved, solved, sizeof(bool), hipMemcpyHostToDevice);

  // ...

  ;

  int *h_perms = new int[perms.size() * no_sums];
  for (int i = 0; i < perms.size(); ++i)
  {
    for (int j = 0; j < 4; ++j)
    {
      h_perms[i * 4 + j] = perms[i][j];
    }
  }
  cout<< perms.size();
  cout<<"number of empty cells"<<no_sums;
  // Allocate device memory for `d_perms`
  int *d_perms;

  hipMalloc(&d_perms, perms.size() * no_sums * sizeof(int));
  hipMemcpy(d_perms, h_perms, perms.size() * no_sums * sizeof(int), hipMemcpyHostToDevice);

  kakuro_kernel<<<grid_dim, block_dim>>>(d_sum_starts_x, d_sum_starts_y, d_sum_ends_x, d_sum_ends_y, d_sum_hints,
                                         d_sum_lengths, d_sum_dirs, d_sol_mat, d_perms, d_t_mats, m, n,
                                         no_sums, d_solved);
  hipDeviceSynchronize();
  // CUDA

  // print_flattened_matrix(d_sol_mat, m, n);
  // TO DO sol_mat_flattened_to_file(mat, d_sol_mat, m, n)
  // Similiar to sol_mat, use hints from mat and values from d_sol_mat

  for (int i = 0; i < n; i++)
  {
    delete mat[i];
    delete sol_mat[i];
  }

  delete mat;
  delete sol_mat;

  delete h_sum_starts_x;
  delete h_sum_starts_y;
  delete h_sum_ends_x;
  delete h_sum_ends_y;
  delete h_sum_hints;
  delete h_sum_lengths;
  delete h_sum_dirs;
  delete h_sol_mat;
  delete[] h_perms;

  hipFree(d_t_mats);
  hipFree(d_sum_starts_x);
  hipFree(d_sum_starts_y);
  hipFree(d_sum_ends_x);
  hipFree(d_sum_ends_y);
  hipFree(d_sum_hints);
  hipFree(d_sum_lengths);
  hipFree(d_sum_dirs);
  hipFree(d_sol_mat);

  return 0;
}
